#include "hip/hip_runtime.h"
#include <iostream>
#include "error.cuh"

// 行主序
#define A(i, j) a[(i) * lda + (j)]
#define B(i, j) b[(i) * ldb + (j)]
#define C(i, j) c[(i) * ldc + (j)]

// A[M][K] B[K][N] C[M][N]
const int M = 128;
const int K = 128;
const int N = 128;

const int ITERATION = 50;

// block_size and grid_size
const dim3 threads_per_block = {16, 16, 1};
const dim3 blocks_per_grid = {M / threads_per_block.x, N / threads_per_block.y, 1};


// 行主序
__global__ void matrixMul0(const float *a, const float *b, float *c, int M, int N, int K)
{
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int lda = K, ldb = N, ldc = N;
    if (col < N && row < M)
    {
        float value = 0;
        for (int k = 0; k < K; k++) 
        {
            value += A(row, k) * B(k, col);
        }
        C(row, col) = value;
    }
}

void test_matrixMat0()
{
    // allocate host for A, B, C
    float *h_a = (float *)malloc(sizeof(float) * M * K);
    float *h_b = (float *)malloc(sizeof(float) * K * N);
    float *h_c = (float *)malloc(sizeof(float) * M * N);

    for (int i = 0; i < M * K; i++)
        h_a[i] = i % 7;
    for (int i = 0; i < K * N; i++)
        h_b[i] = i % 7;

    // test device_c result
    float *test_c = (float *)malloc(sizeof(float) * M * N);
    for (int i = 0; i < M; i ++) {
        for (int j = 0; j < N; j ++) {
            float value = 0;
            for (int k = 0; k < K; k ++) {
                //test_c[i][j] += h_a[i][k] * h_b[k][j]
                value += h_a[i * K + k] * h_b[k * N + j];
            }
            test_c[i * N + j] += value;
        }
    }

    // allocate device for A, B, C
    float *d_a;
    CHECK(hipMalloc((void **)&d_a, sizeof(float) * M * K));
    float *d_b;
    CHECK(hipMalloc((void **)&d_b, sizeof(float) * K * N));
    float *d_c;
    CHECK(hipMalloc((void **)&d_c, sizeof(float) * M * N));

    // copy h_a h_b for d_a d_b
    CHECK(hipMemcpy(d_a, h_a, sizeof(float) * M * K, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(float) * K * N, hipMemcpyHostToDevice));

    // record the time
    float t_sum = 0, t2_sum = 0;
    for (int repeat = 0; repeat <= ITERATION; repeat++)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        // kernel
        matrixMul0<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, M, N, K);

        CHECK(hipMemcpy(h_c, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost)); // 隐式同步

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        } 
        else
        {
            for (int i = 0; i < M * N; i ++) {
                std::cout << i << " h_c : " << h_c[i] << " test_c : " << test_c[i] << std::endl;
            }
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / ITERATION;
    const float t_err = sqrt(t2_sum / ITERATION - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    // Free Memory
    free(h_a);
    free(h_b);
    free(h_c);
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}

int main()
{
    test_matrixMat0();

    return 0;
}
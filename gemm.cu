#include "hip/hip_runtime.h"
#include <iostream>
#include "error.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

// 行主序
#define A(i, j) a[(i) * lda + (j)]
#define B(i, j) b[(i) * ldb + (j)]
#define C(i, j) c[(i) * ldc + (j)]

// A[M][K] B[K][N] C[M][N]
const int M = 1024 * 4;
const int K = 1024 * 4;
const int N = 1024 * 4;

// TILE_WIDTH
const int TILE_M = 16;
const int TILE_K = 16;
const int TILE_N = 16;

const int BLOCK_SIZE = 16;

const int ITERATION = 10;

// block_size and grid_size
const dim3 threads_per_block = {16, 16, 1};
const dim3 blocks_per_grid = {N / threads_per_block.x, M / threads_per_block.y, 1};


// kernel 行主序
__global__ void matrixMul0(const float *a, const float *b, float *c, int M, int N, int K)
{
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int lda = K, ldb = N, ldc = N;
    if (col < N && row < M)
    {
        float value = 0;
        for (int k = 0; k < K; k++) 
        {
            value += A(row, k) * B(k, col);
        }
        C(row, col) = value;
    }
}

// global memory -----> shared memory
__global__ void matrixMul1(const float *a, const float *b, float *c, int M, int N, int K)
{
    __shared__ float Tiled_A[TILE_M][TILE_K];
    __shared__ float Tiled_B[TILE_K][TILE_N];

    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int lda = K, ldb = N, ldc = N;
    if (col < N && row < M)
    {
        float value = 0;
        for (int idx_tile = 0; idx_tile < K / TILE_K; idx_tile ++) 
        {
            // load Tiled_A && Tiled_B from global_memory to shared_memory
            Tiled_A[threadIdx.y][threadIdx.x] = A(row, idx_tile * TILE_K + threadIdx.x);
            Tiled_B[threadIdx.y][threadIdx.x] = B(idx_tile * TILE_K + threadIdx.y, col);    
            __syncthreads(); // sync
            for (int idx_k = 0; idx_k < TILE_K; idx_k ++) 
            {
                value += Tiled_A[threadIdx.y][idx_k] * Tiled_B[idx_k][threadIdx.x];
            }
            __syncthreads();
        }
        C(row, col) = value;
    }
}

__global__ void matrixMul2(const float *a, const float *b, float *c, int M, int N, int K)
{
    __shared__ float Mds[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Nds[BLOCK_SIZE][BLOCK_SIZE];
    float value = 0;
    cooperative_groups::thread_block g = cooperative_groups::this_thread_block();
    int row = g.thread_index().x + g.group_index().x * BLOCK_SIZE;
    int col = g.thread_index().y + g.group_index().y * BLOCK_SIZE;
    int tx = g.thread_index().x;
    int ty = g.thread_index().y;
    for (int i = 0; i < N / BLOCK_SIZE; i ++) 
    {
        Mds[tx][ty] = a[row * K + ty + i * BLOCK_SIZE];
        Nds[tx][ty] = b[col + (tx + i * BLOCK_SIZE) * N];
        g.sync();
        for (int j = 0; j < BLOCK_SIZE; j ++) 
        {
            value += Mds[tx][j] * Nds[j][ty];
            g.sync();
        }
    }
    c[row * N + col] = value;
}

void test_matrixMat()
{
    // allocate host for A, B, C
    float *h_a = (float *)malloc(sizeof(float) * M * K);
    float *h_b = (float *)malloc(sizeof(float) * K * N);
    float *h_cC = (float *)malloc(sizeof(float) * M * N); // cublas
    float *h_c0 = (float *)malloc(sizeof(float) * M * N); // matrixMul0
    float *h_c1 = (float *)malloc(sizeof(float) * M * N); // matrixMul1
    float *h_c2 = (float *)malloc(sizeof(float) * M * N); // matrixMuk2

    for (int i = 0; i < M * K; i++)
        h_a[i] = i % 7;
    for (int i = 0; i < K * N; i++)
        h_b[i] = i % 7;

    // test device_c result
    float *test_c = (float *)malloc(sizeof(float) * M * N);
    for (int i = 0; i < M; i ++) {
        for (int j = 0; j < N; j ++) {
            float value = 0;
            for (int k = 0; k < K; k ++) {
                //test_c[i][j] += h_a[i][k] * h_b[k][j]
                value += h_a[i * K + k] * h_b[k * N + j];
            }
            test_c[i * N + j] += value;
        }
    }

    // allocate device for A, B, C
    float *d_a;
    CHECK(hipMalloc((void **)&d_a, sizeof(float) * M * K));
    float *d_b;
    CHECK(hipMalloc((void **)&d_b, sizeof(float) * K * N));
    float *d_c;
    CHECK(hipMalloc((void **)&d_c, sizeof(float) * M * N));

    // -------------------------- matrix0 -------------------------- //
    // copy h_a h_b for d_a d_b
    CHECK(hipMemcpy(d_a, h_a, sizeof(float) * M * K, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(float) * K * N, hipMemcpyHostToDevice));

    // record the MatrixMul0 time
    float t_sum = 0, t2_sum = 0;
    for (int repeat = 0; repeat <= ITERATION; repeat++)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        // MatrixMul0 kernel
        matrixMul0<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, M, N, K);

        CHECK(hipMemcpy(h_c0, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost)); // 隐式同步

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        } 
        else
        {
            for (int i = 0; i < M * N; i ++) {
                if (abs(h_c0[i] - test_c[i]) > 1e-5) {
                    printf("MatrixMul0 Result Error!\n");
                    break;
                }
            }
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / ITERATION;
    const float t_err = sqrt(t2_sum / ITERATION - t_ave * t_ave);
    printf(" ----- MatrixMul0 ----- \n");
    printf("Time = %g +- %g ms.\n\n", t_ave, t_err);

    // -------------------------- matrix0 -------------------------- //



    // --------------------------  matrix1 -------------------------- //

    // copy h_a h_b for d_a d_b
    CHECK(hipMemcpy(d_a, h_a, sizeof(float) * M * K, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(float) * K * N, hipMemcpyHostToDevice));
    // record the MatrixMul1 time
    t_sum = 0, t2_sum = 0;
    for (int repeat = 0; repeat <= ITERATION; repeat++)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        // MatrixMul0 kernel
        matrixMul1<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, M, N, K);

        CHECK(hipMemcpy(h_cC, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost)); // 隐式同步

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        } 
        else
        {
            for (int i = 0; i < M * N; i ++) {
                if (abs(h_cC[i] - test_c[i]) > 1e-5) {
                    printf("MatrixMul1 Result Error!\n");
                    break;
                }
            }
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave3 = t_sum / ITERATION;
    const float t_err3 = sqrt(t2_sum / ITERATION - t_ave3 * t_ave3);
    printf(" ----- MatrixMul1 ----- \n");
    printf("Time = %g +- %g ms.\n\n", t_ave3, t_err3);

    // --------------------------  matrix1 -------------------------- //





    // -------------------------- cublas -------------------------- //
    // copy h_a h_b for d_a d_b
    CHECK(hipMemcpy(d_a, h_a, sizeof(float) * M * K, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(float) * K * N, hipMemcpyHostToDevice));
    // record the cublas time
    t_sum = 0, t2_sum = 0;
    for (int repeat = 0; repeat < ITERATION; repeat ++) {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        // cublas kernel
        hipblasHandle_t blas_handle;
        hipblasCreate(&blas_handle);
        float alpha = 1.0f, beta = 0.0f;
        // CHECK(hipMemcpy(d_c, h_c, sizeof(float) * M * N, hipMemcpyHostToDevice));
        hipblasSgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, K, d_b, N, &beta, d_c, N);

        CHECK(hipMemcpy(h_c1, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost)); // 隐式同步

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        } 
        else
        {
            for (int i = 0; i < M * N; i ++) {
                if (abs(h_c1[i] - test_c[i]) > 1e-5) {
                    printf("cublas Result Error!\n");
                    break;
                }
            }
            // for (int i = 0; i < M * N; i ++) {
            //     std::cout << i << " h_c1 " << h_c1[i] << " test_c " << test_c[i] << std::endl;
            // }
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));

    }

    const float t_ave2 = t_sum / ITERATION;
    const float t_err2 = sqrt(t2_sum / ITERATION - t_ave * t_ave);
    printf(" ----- cublas ----- \n");
    printf("Time = %g +- %g ms.\n\n", t_ave2, t_err2);

    // -------------------------- cublas -------------------------- //




    // --------------------------  matrix2 -------------------------- //

    // copy h_a h_b for d_a d_b
    CHECK(hipMemcpy(d_a, h_a, sizeof(float) * M * K, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(float) * K * N, hipMemcpyHostToDevice));
    // record the MatrixMul1 time
    t_sum = 0, t2_sum = 0;
    for (int repeat = 0; repeat <= ITERATION; repeat++)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        // MatrixMul0 kernel
        matrixMul2<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, M, N, K);

        CHECK(hipMemcpy(h_c2, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost)); // 隐式同步

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        } 
        else
        {
            for (int i = 0; i < M * N; i ++) {
                if (abs(h_c2[i] - test_c[i]) > 1e-5) {
                    printf("MatrixMul1 Result Error!\n");
                    break;
                }
            }
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave4 = t_sum / ITERATION;
    const float t_err4 = sqrt(t2_sum / ITERATION - t_ave4 * t_ave4);
    printf(" ----- MatrixMul2 ----- \n");
    printf("Time = %g +- %g ms.\n\n", t_ave4, t_err4);

    // --------------------------  matrix2 -------------------------- //


    // Free Memory
    free(h_a);
    free(h_b);
    free(h_cC);
    free(h_c0);
    free(h_c1);
    free(h_c2);
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}



int main()
{
    test_matrixMat();

    return 0;
}